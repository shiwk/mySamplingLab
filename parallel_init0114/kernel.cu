#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\transform_reduce.h>
#include <utility>
#include <ctime>
#include <hip/device_functions.h>
#include <stdio.h>
#include <vector>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/tuple.h>
#include <fstream>

using namespace std;

using namespace thrust;

template <typename T>
struct square
{
	__host__ __device__
		T operator()(const T& x) const {
		return x * x;
	}
};

struct prg
{
	float a, b;

	__host__ __device__
		prg(float _a, float _b) : a(_a), b(_b) {};

	__host__ __device__
		float operator()(const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<float> dist(a, b);
		rng.discard(n);

		return dist(rng);
	}
};


struct Point
{
	float x;
	float y;
	Point() :x(0.0f), y(0.0f) {}

	__host__ __device__
		Point(float h, float w) : x(h), y(w) {}
};



//__shared__ PointTuple *raw_ptr;

__constant__ int wNum[1];
__constant__ int hNum[1];
__constant__ float width[1];
__constant__ float radius[1];//R
__constant__ int cinit[1];//³õÊ¼»¯µãÊýÁ¿
#define M_PI 3.141592654f

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t init_point(thrust::device_ptr<Point>, unsigned int);
hipError_t randomInsertPoints(unsigned int);


__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = blockIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void testPointer(Point *raw_ptr) {
	//	cout << "pointer[0]" << *a<<endl;

	extern __shared__ Point dev_zip[];
	extern __shared__ int sum[];
	sum[0] = 0;
	int j = threadIdx.x;
	int i = threadIdx.x + blockDim.x*blockIdx.x;

	if (i < 300000) dev_zip[j] = raw_ptr[i];

	__syncthreads();



	atomicAdd(&sum[0], 1);
}



__global__ void testNum(int *raw_ptr) {

	/*extern __shared__ int share_nums[];
	extern __shared__ int sum[];
	sum[0] = 0;
	int j = threadIdx.x;
	for (int i = j; i < 30; i += (512))
	{
	share_nums[i] = raw_ptr[i];
	}

	__syncthreads();


	int num = 0;
	for (; j < 30; j += (512))
	{
	if (share_nums[j])
	num++;
	}
	atomicAdd(&sum[0], num);*/

	extern __shared__ int sm[];

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.x;
	if (i<300000) sm[j] = raw_ptr[i];

	__syncthreads();


}




__global__ void addWithSM(int *c, const int *a) {
	int i = threadIdx.x;
	extern __shared__ int seme[];
	seme[i] = a[i];
	__syncthreads();

	if (i == 0) {
		c[0] = 0;
		for (int d = 0; d < 5; d++) {
			c[0] += seme[d] * seme[d];
		}
	}
	if (i == 1) {
		c[1] = 0;
		for (int d = 0; d < 5; d++) {
			c[i] += seme[d];
		}
	}
	if (i == 2)
	{
		c[i] = 1;
		for (int d = 0; d < 5; d++) {
			c[2] *= seme[d];
		}
	}

}

__global__ void addKernel_thd(int *c, int *a, int *b) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}


__global__ void initpoints(float *h_ptr, float *w_ptr, Point *init_ptr) {
	int i = blockIdx.x;
	int j = threadIdx.x;
	int index = i*blockDim.x + j;
	if (index < cinit[0]) {
		init_ptr[index] = Point(h_ptr[index], w_ptr[index]);
	}
}




__global__ void insertPointsErase(Point *raw_ptr, bool *erase_ptr, float *ran_ptr, Point *points_ptr, bool *flag_ptr, int *neis_ptr) {


	extern __shared__ Point p[];
	extern __shared__ int save[];
	extern __shared__ int n[];
	//extern __shared__ int value[];

	//thrust::default_random_engine rng;
	//thrust::uniform_real_distribution<float> range(0.0f, 1000.0f);
	//rng.discard(i);

	int i = blockIdx.x;
	int j = threadIdx.x;



	//if (i == j){
	//	ran_ptr[i] = range(rng);
	//	//int index = ran_ptr[i] / width[0];
	//}
	p[j] = raw_ptr[j];

	__syncthreads();

	if (i != j) {
		float dis = sqrt(pow(p[j].x - p[i].x, 2) + pow(p[j].y - p[i].y, 2));
		if (dis < radius[0]) {
			if (ran_ptr[i] < ran_ptr[j] && !erase_ptr[j]) {
				erase_ptr[i] = true;
			}
			//erase_ptr[i] = true;
		}
	}

	if (j < 6) {
		if (j == 0) n[0] = 0;
		save[j] = -1;
		//value[j] = -1;
	}
	__syncthreads();



	if (!erase_ptr[i]) {
		/*int x = p[i].x / width[0];
		int y = p[i].y / width[0];

		int index = x*wNum[0]+y;*/

		for (size_t k = j; k < cinit[0]; k += 512)
		{
			/*int hi = k / wNum[0];
			int wi = k - hi*wNum[0];
			if ((hi - x <= 2 && hi - x>-2) && (wi - y <= 2&&wi-y>=-2)){

			}*/
			float dX = points_ptr[k].x - p[i].x;
			float dY = points_ptr[k].y - p[i].y;
			float dis = sqrt(pow(dX, 2) + pow(dY, 2));

			if (dis<radius[0] && !flag_ptr[k]) {


				atomicAdd(&n[0], 1);

				float vectorA[] = { -1, 0 };
				float vectorB[] = { dX, dY };


				double dotProduct = -1 * dX;
				double cos = dotProduct / (1 * sqrt(pow(dX, 2) + pow(dY, 2)));



				double res = acos(cos);
				res = dY > 0 ? res : 2 * M_PI - res;

				int s = floor(res / (M_PI / 3));//·ÖÇø
				s = s > 5 ? 5 : s;
				save[s] = k;
				flag_ptr[k] = true;
			}
		}
	}
	__syncthreads();
	if (j<6) {
		if (save[j] != -1) flag_ptr[save[j]] = false;
	}
	if (i == j) {
		neis_ptr[j] = n[0];
		if (n[0] == 1) {
			for (size_t t = 0; t < 6; t++) {
				if (save[t] != -1) points_ptr[save[t]] = raw_ptr[i];
			}
		}
	}

}




float window_width = 350.0;
float window_height = 350.0;

const size_t N = 1000;

const unsigned int numRandom = 300000;
int init[1];

float ratio = 0.75;
float R[1];//¶¨Òå±ê×¼µã¾à




		   //float u = 1.0;//¶¨Òåµ¥Î»¾àÀë



float d[1]; //¸ñ×Ó±ß³¤


int h[1];
int w[1];

int main()
{
	ratio = 0.75;
	float D = 2 * sqrt(window_width*window_height / (2 * N*sqrt(3)));
	R[0] = ratio * D;
	cout << R[0] << endl;
	d[0] = sqrt(2)*R[0] / 2;

	h[0] = ceil(window_height / d[0]);
	w[0] = ceil(window_width / d[0]);
	init[0] = numRandom;
	//copy host  data to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(hNum), h, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(wNum), w, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(width), d, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(radius), R, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(cinit), init, sizeof(int));


	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;
	cudaStatus = hipGetDeviceCount(&num);

	for (int i = 0; i<num; i++)
	{
		hipGetDeviceProperties(&prop, i);
	}




	cudaStatus = randomInsertPoints(512);
	// Add vectors in parallel.
	// cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "compute failed!");
		return 1;
	}


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	/*for (int i = 0; i<arraySize; i++)
	{
	if (c[i] != (a[i] + b[i]))
	{
	printf("Error in %d\n", i);
	}
	}*/
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}



	//time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int i = 0; i < 1000; i++) {
		addKernel_thd << <1, 512 >> >(dev_c, dev_a, dev_b);

	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float tm;
	hipEventElapsedTime(&tm, start, stop);
	printf("GPU ll time:%.6f ms.\n", tm);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}





hipError_t randomInsertPoints(const unsigned int size)
{

	hipError_t cudaStatus;
	bool *erase_ptr;


	thrust::device_ptr<Point> dev_init = thrust::device_malloc<Point>(numRandom);

	cudaStatus = init_point(dev_init, numRandom);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init points error!\n", cudaStatus);
		return cudaStatus;
		//goto Errors;
	}


	Point *points_ptr = thrust::raw_pointer_cast(dev_init);

	/*for (size_t i = 0; i < numRandom; i+=10000)
	{
	cout << points_ptr[i].x << " " << points_ptr[i].y << endl;
	}*/
	//random insert points set




	hipMalloc((void**)&erase_ptr, size * sizeof(bool));
	thrust::device_ptr<bool> dev_erase_ptr(erase_ptr);



	thrust::device_ptr<bool>dev_flag_ptr = thrust::device_malloc<bool>(numRandom);
	thrust::fill(dev_flag_ptr, dev_flag_ptr + numRandom, false);
	bool *flag_ptr = thrust::raw_pointer_cast(dev_flag_ptr);



	thrust::device_vector<float> x(size);
	thrust::device_vector<float> y(size);

	//threads'random value sets
	thrust::device_vector<float> ran(size);

	//threads' neighbor points to erase
	//thrust::device_vector<int> neis(size);
	thrust::device_ptr<int>dev_nei_ptr = thrust::device_malloc<int>(size);
	int *neis_ptr = thrust::raw_pointer_cast(dev_nei_ptr);


	int times = 0;

	int flagnum = 0;
	bool flag[numRandom];
	Point *points;
	points = (Point*)malloc(numRandom * sizeof(Point));

	string file = "parallel";
	

	ofstream outfile;


	while (true)
	{
		times++;

		thrust::fill(dev_erase_ptr, dev_erase_ptr + size, false);

		thrust::fill(dev_nei_ptr, dev_nei_ptr + size, -1);

		prg rx = prg(0.0f, window_height);
		unsigned int offset = time(NULL);
		thrust::counting_iterator<unsigned int> index_sequence_beginx(offset), index_sequence_beginy(offset + times);
		thrust::transform(index_sequence_beginx,
			index_sequence_beginx + size,
			x.begin(),
			rx);

		prg ry = prg(0.0f, window_width);
		thrust::transform(index_sequence_beginy,
			index_sequence_beginy + size,
			y.begin(),
			ry);




		thrust::counting_iterator<unsigned int> index_sequence_begin_ran(offset + 1000 + times);
		prg rran = prg(0.0f, 1000.0f);
		thrust::transform(index_sequence_begin_ran,
			index_sequence_begin_ran + size,
			ran.begin(),
			rran);


		float *ran_ptr = thrust::raw_pointer_cast(ran.data());

		thrust::device_vector<Point> dev_tuvec;

		for (int i = 0; i < size; i++) {
			//cout << x[i] << " " << y[i]<<endl;
			int xIndex = x[i] / d[0];
			int yIndex = y[i] / d[0];
			if (xIndex >= h[0] || yIndex >= w[0]) {
				cout << h[0] << " " << w[0] << " " << d[0] << endl;
				std::system("pause");
			}
			dev_tuvec.push_back(Point(x[i], y[i]));

		}


		Point *raw_ptr = thrust::raw_pointer_cast(dev_tuvec.data());





		//time
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		insertPointsErase << < size, size, size * sizeof(Point) + 7 * sizeof(int), 0 >> >(raw_ptr, erase_ptr, ran_ptr, points_ptr, flag_ptr, neis_ptr);


		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float tm;
		hipEventElapsedTime(&tm, start, stop);
		printf("GPU Elapsed time:%.6f ms.\n", tm);


		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Errors;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Errors;
		}



		bool erase[512];
		cudaStatus = hipMemcpy(erase, erase_ptr, size * sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Errors;
		}


		cudaStatus = hipMemcpy(flag, flag_ptr, numRandom * sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Errors;
		}



		cudaStatus = hipMemcpy(points, points_ptr, numRandom * sizeof(Point), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Errors;
		}
		// copy memory to a new device_vector (which automatically allocates memory)
		//thrust::device_vector<bool> vec(dev_erase_ptr, dev_erase_ptr + size);
		/*int eraseNum = 0;
		for (int i = 0; i < size; i++){
		if (erase[i]) eraseNum++;
		}
		cout << "erase£º" << eraseNum << endl;
		*/


		int eraseNum = 0;
		int neis = 0;
		for (int i = 0; i < size; i++) {
			if (dev_erase_ptr[i]) eraseNum++;
			if (dev_nei_ptr[i] == 1) neis++;
		}
		cout << "erase" << eraseNum << endl;
		cout << "only one neighbors" << neis << endl;


		flagnum = 0;
		for (size_t i = 0; i < numRandom; i++)
		{
			if (flag[i]) {
				flagnum++;
				//cout << i << endl;
			}
		}
		cout << times << " " << flagnum << endl;
		if (flagnum >= numRandom - 1000) break;

	}
	file.append(to_string(flagnum));
	file.append(".txt");

	outfile.open(file);
	outfile << numRandom - flagnum << endl;
	for (size_t i = 0; i < numRandom; i++)
	{
		if (!flag[i]) {
			//cout << i << endl;
			outfile << points[i].x / window_height << " " << points[i].y / window_width << endl;
		}
	}
	outfile.close();

Errors:
	hipFree(erase_ptr);


	return cudaStatus;
}



hipError_t init_point(thrust::device_ptr<Point>dev_init, unsigned int size) {



	hipError_t cudaStatus;




	//random points init
	//thrust::device_vector<Point> initpoints;
	unsigned int off = time(NULL);
	thrust::counting_iterator<unsigned int> index_sequence_begin_h(off), index_sequence_begin_w(off + 1);
	thrust::device_vector<float> h(size);
	thrust::device_vector<float> w(size);

	prg rh = prg(0.0f, window_height);
	thrust::transform(index_sequence_begin_h,
		index_sequence_begin_h + size,
		h.begin(),
		rh);

	prg rw = prg(0.0f, window_width);
	thrust::transform(index_sequence_begin_w,
		index_sequence_begin_w + size,
		w.begin(),
		rw);


	hipEvent_t start_host, stop_host;
	hipEventCreate(&start_host);
	hipEventCreate(&stop_host);
	hipEventRecord(start_host, 0);




	float *h_ptr = thrust::raw_pointer_cast(h.data());
	float *w_ptr = thrust::raw_pointer_cast(w.data());



	//thrust::device_ptr<Point> dev_init = thrust::device_malloc<Point>(numRandom);
	Point *init_ptr = thrust::raw_pointer_cast(dev_init);

	size_t block = ceil(size / 512);
	initpoints << <block, 512 >> >(h_ptr, w_ptr, init_ptr);



	hipEventRecord(stop_host, 0);
	hipEventSynchronize(stop_host);
	float tm_host;
	hipEventElapsedTime(&tm_host, start_host, stop_host);
	printf("Cpu Elapsed time:%.6f ms.\n", tm_host);




	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init error! hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init error! launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init error! hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

		return cudaStatus;
	}

	return cudaStatus;
}